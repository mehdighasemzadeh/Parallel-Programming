#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

__global__ void bit_reversal(float* input , float* output,  unsigned int N)
{   
    int thread_id = (threadIdx.x + blockIdx.x * blockDim.x) + (blockIdx.y*(1024*16384));
 
    float log2N = log2f(N);
    int n = 0;
    int  x = thread_id;
    for (int i = 0; i < log2N; i++)
    {
        n <<= 1;
        n |= (x & 1);
        x >>= 1;
    }

      output[thread_id] = input[n];
    __syncthreads();
}












__global__ void fft(float *x_r , float *x_i ,float *rev_x_r, float *rev_x_i, int N, int stage, int butterfly_width, int step){

	const double twoPIdivN = 2 * PI / N;
	int thread_id = (threadIdx.x + blockIdx.x * blockDim.x) + (blockIdx.y*(1024*16384));
	//======= temp variable========= 
	float wn_r , wn_i;
	float temp1_r , temp1_i ;
	float temp2_r , temp2_i ;
  
	//=======bit r ===============
	/*
	if(stage==1){
	  int r = bit_reversal(thread_id, N);
	  rev_x_r[thread_id] = x_r[r];
	  __syncthreads();
	}
  
  
	if(stage==1){
	  int r = bit_reversal(thread_id, N);
	  rev_x_i[thread_id] = x_i[r];
	  __syncthreads();
	}
	*/
  
  
  
	//============== fft parametr ===============
	int pos = thread_id / butterfly_width * step;
	int j = thread_id % butterfly_width;
	int res = pos + j;
	if (res < N){
		
	  //Wn = e^(-j*2*PI/N) converted with euler's formula(real and imaginary parts)
	  wn_r =  cos(twoPIdivN * j * N / step);
	  wn_i = -sin(twoPIdivN * j * N / step);
  
	  temp1_r = rev_x_r[res];
	  temp1_i = rev_x_i[res];
	  temp2_r = rev_x_r[res + butterfly_width] * wn_r - rev_x_i[res + butterfly_width] * wn_i;
	  temp2_i = rev_x_i[res + butterfly_width] * wn_r + rev_x_r[res + butterfly_width] * wn_i;
  
	  rev_x_r[res]                   = temp1_r + temp2_r;
	  rev_x_i[res]                   = temp1_i + temp2_i;
	  rev_x_r[res + butterfly_width] = temp1_r - temp2_r;
	  rev_x_i[res + butterfly_width] = temp1_i - temp2_i;
		  
	  __syncthreads();
	}
	//===========end if ====================
  
}







void fft_caller(float *x_r , float *x_i , float *rev_x_r , float *rev_x_i , int N)
{

	dim3 blocks(16384,4);
	dim3 threadsPerBlock(1024,1);
 /*
    if (N>=1024 && N< 67108864 ){
      dim3 blocks(N/1024,1);
	  dim3 threadsPerBlock(1024,1);
    }

    if (N==67108864){
     	dim3 blocks(32768,2);
	    dim3 threadsPerBlock(1024,1);
    }
	*/

  	float stages = log2f(N);
  	int butterfly_width, step;

  	if (N > 1)
  	{
    	for (int stage = 1; stage <= stages; stage++)
    {   
        //printf("%d ", stage);
        step = 1 << stage;
        butterfly_width = step >> 1;
        fft<<<blocks, threadsPerBlock>>>(x_r , x_i , rev_x_r, rev_x_i, N, stage, butterfly_width, step);
    }
 
  }
}

  










//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, /*float* X_r_d, float* X_i_d,*/ const unsigned int N, const unsigned int M)
{
	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.
	
	// set thread count
	dim3 blocks(16384,4);
	dim3 threadsPerBlock(1024,1);
	/*
    if (N>=1024 && N< 67108864 ){
      dim3 blocks(N/1024,1);
	  dim3 threadsPerBlock(1024,1);
    }

    if (N==67108864){
     	dim3 blocks(32768,2);
	    dim3 threadsPerBlock(1024,1);
    }

	*/

	float* rev_x_r;
	float* rev_x_i;


	hipMalloc((void**)&rev_x_r, N * sizeof(float));
	bit_reversal<<<blocks, threadsPerBlock>>>(x_r_d , rev_x_r, N);
	hipMemcpy(x_r_d, rev_x_r , N * sizeof(float), hipMemcpyDeviceToDevice);
	hipFree(rev_x_r);


	hipMalloc((void**)&rev_x_i, N * sizeof(float));
	bit_reversal<<<blocks, threadsPerBlock>>>(x_i_d , rev_x_i, N);
	hipMemcpy(x_i_d, rev_x_i , N * sizeof(float), hipMemcpyDeviceToDevice);
	hipFree(rev_x_i);


	fft_caller(rev_x_r , rev_x_i , x_r_d ,x_i_d , N);
	
	



	
}
