#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "fft.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

__device__ int bit_reversal(unsigned int N , unsigned int M , int i)
{   
    int thread_id = i;
    int log2N = M;
    int n = 0;
    int  x = thread_id;
    for (int i = 0; i < log2N; i++)
    {
        n <<= 1;
        n |= (x & 1);
        x >>= 1;
    }
    return n ; 
}


__global__ void bit_reversal_helper(float* inputr,float* inputi , unsigned int N , unsigned int M ){
  int thread_id = (threadIdx.x + blockIdx.x * blockDim.x) + (blockIdx.y*(1024*32768));
  int j ;
  float temp1 ;
  float temp2 ; 
  int i =  thread_id ; 
  j = bit_reversal(N, M,i);
    if (i<j){
      temp1 = inputr[i] ; 
      inputr[i] = inputr[j];
      inputr[j] = temp1 ;

      temp2 = inputi[i] ; 
      inputi[i] = inputi[j];
      inputi[j] = temp2 ;

    }
}





__global__ void fft(float *rev_x_r, float *rev_x_i, int N, int stage, int butterfly_width, int step){

	const double twoPIdivN = 2 * PI / N;
	int thread_id = (threadIdx.x + blockIdx.x * blockDim.x) + (blockIdx.y*(512*32768));
	//======= temp variable========= 
	float wn_r , wn_i;
	float temp1_r , temp1_i ;
	float temp2_r , temp2_i ;
  
	//============== fft parametr ===============
	int pos = (thread_id / butterfly_width) * step;
	int j = thread_id % butterfly_width;
	int res = pos + j;
	if (res < N){
		
	  wn_r =  cos(twoPIdivN * j * N / step);
	  wn_i = -sin(twoPIdivN * j * N / step);
  
	  temp1_r = rev_x_r[res];
	  temp1_i = rev_x_i[res];
	  temp2_r = rev_x_r[res + butterfly_width] * wn_r - rev_x_i[res + butterfly_width] * wn_i;
	  temp2_i = rev_x_i[res + butterfly_width] * wn_r + rev_x_r[res + butterfly_width] * wn_i;
  
	  rev_x_r[res]                   = temp1_r + temp2_r;
	  rev_x_i[res]                   = temp1_i + temp2_i;
	  rev_x_r[res + butterfly_width] = temp1_r - temp2_r;
	  rev_x_i[res + butterfly_width] = temp1_i - temp2_i;
		  
	  //__syncthreads();
	}
	//===========end if ====================
  
}













__global__ void fft_SM( float *rev_x_rg, float *rev_x_ig, int N){

	const double twoPIdivN = 2 * PI / N;
	int thread_id = (threadIdx.x + blockIdx.x * blockDim.x) + (blockIdx.y*(512*32768));
  __shared__ float rev_x_r[1024];
  __shared__ float rev_x_i[1024];
  rev_x_r[2*tx] = rev_x_rg[2*thread_id];
  rev_x_r[2*tx+1] = rev_x_rg[2*thread_id+1];
  rev_x_i[2*tx] = rev_x_ig[2*thread_id];
  rev_x_i[2*tx+1] = rev_x_ig[2*thread_id+1];
  __syncthreads();

	//======= temp variable=========
  float wn_r , wn_i;
	float temp1_r , temp1_i ;
	float temp2_r , temp2_i ;

  int step;
  int stages = 10;
  int butterfly_width;
  for (int stage = 1; stage <= stages; stage++)
  {   
    step = 1 << stage;
    butterfly_width = step >> 1; 

  
  
	//============== fft parametr ===============
	int pos = (tx / butterfly_width) * step;
	int j = tx % butterfly_width;
	int res = pos + j;
	if (res < N){
		
	  wn_r =  cos(twoPIdivN * j * N / step);
	  wn_i = -sin(twoPIdivN * j * N / step);
  
	  temp1_r = rev_x_r[res];
	  temp1_i = rev_x_i[res];
	  temp2_r = rev_x_r[res + butterfly_width] * wn_r - rev_x_i[res + butterfly_width] * wn_i;
	  temp2_i = rev_x_i[res + butterfly_width] * wn_r + rev_x_r[res + butterfly_width] * wn_i;
  
	  rev_x_r[res]                   = temp1_r + temp2_r;
	  rev_x_i[res]                   = temp1_i + temp2_i;
	  rev_x_r[res + butterfly_width] = temp1_r - temp2_r;
	  rev_x_i[res + butterfly_width] = temp1_i - temp2_i;
		  
	__syncthreads();

  }
	}

  rev_x_rg[2*thread_id] = rev_x_r[2*tx];
  rev_x_rg[2*thread_id+1] = rev_x_r[2*tx+1];
  rev_x_ig[2*thread_id] = rev_x_i[2*tx];
  rev_x_ig[2*thread_id+1] = rev_x_i[2*tx+1];
	//===========end if ====================


  
}







void fft_helper(float *rev_x_r , float *rev_x_i , int N , unsigned int M)
{
	  dim3 blocks;
	  dim3 threadsPerBlock;

    if(N<=1024){
    blocks.x = 1 ;
	  blocks.y = 1 ;
    threadsPerBlock.x = N/2;
    threadsPerBlock.y = 1 ;
    }
    if (N>1024 && N< 67108864 ){
    blocks.x = N/1024 ;
	  blocks.y = 1 ;
    threadsPerBlock.x = 512;
    threadsPerBlock.y = 1 ;
    }

    if (N==67108864){
    blocks.x = 32768 ;
	  blocks.y = 2 ;
    threadsPerBlock.x = 512;
    threadsPerBlock.y = 1 ;
    }
	
    //======== 10 frist stage =======
    fft_SM<<<blocks, threadsPerBlock>>>(rev_x_r, rev_x_i, N);
   //============stage 11 to end================================
    int stages = M;
  	int butterfly_width, step;

  	if (N > 1)
  	{
    	for (int stage = 11; stage <= stages; stage++)
    {   
        step = 1 << stage;
        butterfly_width = step >> 1;
        fft<<<blocks, threadsPerBlock>>>(rev_x_r, rev_x_i, N, stage, butterfly_width, step);
    }
 
  }

}

  










//-----------------------------------------------------------------------------
void gpuKernel(float* x_r_d, float* x_i_d, /*float* X_r_d, float* X_i_d,*/ const unsigned int N, const unsigned int M)
{
	// In this function, both inputs and outputs are on GPU.
	// No need for hipMalloc, hipMemcpy or hipFree.
	
	// set thread count
    dim3 blocks;
	  dim3 threadsPerBlock;
    if(N<1024){
    blocks.x = 1 ;
	  blocks.y = 1 ;
    threadsPerBlock.x = N;
    threadsPerBlock.y = 1 ;
  }
    if (N>=1024 && N< 67108864 ){
    blocks.x = N/1024 ;
	  blocks.y = 1 ;
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1 ;
    }

    if (N==67108864){
    blocks.x = 32768 ;
	  blocks.y = 2 ;
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1 ;
    }

	

bit_reversal_helper<<<blocks, threadsPerBlock>>>(x_r_d , x_i_d , N , M);
fft_helper(x_r_d ,x_i_d , N ,M);
	
	



	
}
