#include "hip/hip_runtime.h"
//ONLY MODIFY THIS FILE!
//YOU CAN MODIFY EVERYTHING IN THIS FILE!

#include "bmm.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// TILEX and TILEY are used to set the number of threads in a CUDA block 
#define TILEX 32
#define TILEY 16

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

dim3 getDimGrid(const int m, const int n) {
	dim3 dimGrid(n/TILEX,n/TILEY);
	return dimGrid;
}
dim3 getDimBlock(const int m, const int n) {
	dim3 dimBlock(TILEX,TILEY);
	return dimBlock;
}
__global__ void kernelFunc(float* ad, float* bd, float* cd, const int m, const int n) {

	// write your GPU kernel function here



	if (TILEX > TILEY){
    __shared__ float As[TILEY][TILEY];
  	__shared__ float Bs[TILEY ][TILEX];
 
  	int x = threadIdx.x + blockIdx.x * blockDim.x; 
  	int y = threadIdx.y + blockIdx.y * blockDim.y; 

  	float Pvalue = 0;
  	for (int m1 = 0; m1 < n/TILEY; m1++){
    		if(threadIdx.x<TILEY){	 
    		As[threadIdx.y][threadIdx.x] = ad[y * n + (m1 * TILEY + threadIdx.x)]; 
    		}

    		Bs[threadIdx.y][threadIdx.x] = bd[(m1 * TILEY + threadIdx.y) * n + x];
   	 	__syncthreads();


    		for (int k = 0; k < TILEY; k++)
      		Pvalue += As[threadIdx.y][k] * Bs[k][threadIdx.x];
   		__syncthreads();
  	}
  
  // write back to the global memory
  	cd[y * n + x] = Pvalue;   
  } 
    


// TILEX < TILEY :
    
   if (TILEX < TILEY){
    __shared__ float As[TILEY][TILEX];
  	__shared__ float Bs[TILEX ][TILEX];
 
  	int x = threadIdx.x + blockIdx.x * blockDim.x; 
  	int y = threadIdx.y + blockIdx.y * blockDim.y; 

  	float Pvalue = 0;
  	for (int m1 = 0; m1 < n/TILEX; m1++){
    			 
    		As[threadIdx.y][threadIdx.x] = ad[y * n + (m1 * TILEX + threadIdx.x)]; 
    		
      if(threadIdx.y<TILEX){
    	  Bs[threadIdx.y][threadIdx.x] = bd[(m1 * TILEX + threadIdx.y) * n + x];
      }
   	 	__syncthreads();


    		for (int k = 0; k < TILEX; k++)
      		Pvalue += As[threadIdx.y][k] * Bs[k][threadIdx.x];
   		__syncthreads();
  	}
  
  // write back to the global memory
  	cd[y * n + x] = Pvalue;   
  }






  // TILEX == TILEY :
    
   if (TILEX == TILEY){
    __shared__ float As[TILEY][TILEX];
  	__shared__ float Bs[TILEY][TILEX];
 
  	int x = threadIdx.x + blockIdx.x * blockDim.x; 
  	int y = threadIdx.y + blockIdx.y * blockDim.y; 

  	float Pvalue = 0;
  	for (int m1 = 0; m1 < n/TILEY; m1++){
    			 
    		As[threadIdx.y][threadIdx.x] = ad[y * n + (m1 * TILEY + threadIdx.x)]; 
    	  Bs[threadIdx.y][threadIdx.x] = bd[(m1 * TILEX + threadIdx.y) * n + x];
      
   	 	__syncthreads();

    		for (int k = 0; k < TILEX; k++)
      		Pvalue += As[threadIdx.y][k] * Bs[k][threadIdx.x];
   		__syncthreads();
  	}
  
  // write back to the global memory
  	cd[y * n + x] = Pvalue;   
  }





}
