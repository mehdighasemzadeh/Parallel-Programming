#include "hip/hip_runtime.h"
// ONLY MODIFY THIS FILE

#include "scan2.h"
#include "gpuerrors.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

// you may define other parameters here!
// you may define other macros here!
// you may define other functions here!

__global__ void scanl(float* ad, float* cd ,  float* cqd){
		int n =1024;
		__shared__ float As[1024];
  	__shared__ float Cs[1024];
		int offset = 1;
		int x = (threadIdx.x + blockIdx.x * blockDim.x) + (blockIdx.y*(1024*32768));

		As[threadIdx.x] = ad[x];
		// reduce part---------
		for (int d=512 ; d > 0 ; d = d/2){
			__syncthreads() ;
			if ( threadIdx.x < d){
				int ai = offset * (2*  threadIdx.x +1) -1;
				int bi = offset * (2*  threadIdx.x +2) -1;
				As[bi] += As [ai];
			}
			offset *= 2;
			}
		// end reduce part
		
		
		
		if ( threadIdx.x == 0){
				Cs[threadIdx.x + n - 1] = As [threadIdx.x + n - 1] ;
				As [threadIdx.x + n - 1] = 0 ; 
		}


		for (int d = 1; d < n; d *= 2){
				offset /= 2 ;
				__syncthreads() ;
				if ( threadIdx.x < d){
					int ai = offset * (2* threadIdx.x +1) -1;
					int bi = offset * (2* threadIdx.x +2) -1;
					float t = As[ai ];
					As[ai] = As[bi ];
					As[bi] += t;
				}
		}

		__syncthreads () ;

		if (threadIdx.x >=1){
				Cs[threadIdx.x - 1] = As[threadIdx.x] ;
		}
		__syncthreads () ;
		cd[x] = Cs[threadIdx.x];
		if(threadIdx.x == 0 ){
		cqd[blockIdx.x + 32768*blockIdx.y ] = Cs[threadIdx.x + 1023];
		}

			
}




__global__ void add(float* cd ,  float* cqd){
		int x = (threadIdx.x + blockIdx.x * blockDim.x) + (blockIdx.y*(1024*32768));
		if (blockIdx.x + 32768*blockIdx.y>0){
		cd[x] = cd[x] + cqd[blockIdx.x + 32768*blockIdx.y -1];
		}
 		
}



__global__ void scan_s(float* ad, float* cd ,const int n ){
		int x = threadIdx.x;
		int i ;
		cd[x] = ad[x];
		for(i=1 ; i<n ;i++){
				cd[i+x] = cd[i+x -1] + ad[i];
		}

		
}




__global__ void add_l(float* cd ,  const float back){
		int x = (threadIdx.x + blockIdx.x * blockDim.x) + (blockIdx.y*(1024*32768));
		cd[x] += back ; 
		}





__global__ void scan128(float* ad, float* cd){
		int n =128;
		__shared__ float As[128];
  	__shared__ float Cs[128];
		int offset = 1;
		int x = threadIdx.x + blockIdx.x * blockDim.x;

		As[threadIdx.x] = ad[x];
		// reduce part---------
		for (int d=64 ; d > 0 ; d = d/2){
			__syncthreads() ;
			if ( threadIdx.x < d){
				int ai = offset * (2*  threadIdx.x +1) -1;
				int bi = offset * (2*  threadIdx.x +2) -1;
				As[bi] += As [ai];
			}
			offset *= 2;
			}
		// end reduce part
		
		
		
		if ( threadIdx.x == 0){
				Cs[threadIdx.x + n - 1] = As [threadIdx.x + n - 1] ;
				As [threadIdx.x + n - 1] = 0 ; 
		}


		for (int d = 1; d < n; d *= 2){
				offset /= 2 ;
				__syncthreads() ;
				if ( threadIdx.x < d){
					int ai = offset * (2* threadIdx.x +1) -1;
					int bi = offset * (2* threadIdx.x +2) -1;
					float t = As[ai ];
					As[ai] = As[bi ];
					As[bi] += t;
				}
		}

		__syncthreads () ;

		if (threadIdx.x >=1){
				Cs[threadIdx.x - 1] = As[threadIdx.x] ;
		}
		__syncthreads () ;
		cd[x] = Cs[threadIdx.x];			
}









__global__ void scan64(float* ad, float* cd){
		int n =64;
		__shared__ float As[64];
  	__shared__ float Cs[64];
		int offset = 1;
		int x = threadIdx.x + blockIdx.x * blockDim.x;

		As[threadIdx.x] = ad[x];
		// reduce part---------
		for (int d=32 ; d > 0 ; d = d/2){
			__syncthreads() ;
			if ( threadIdx.x < d){
				int ai = offset * (2*  threadIdx.x +1) -1;
				int bi = offset * (2*  threadIdx.x +2) -1;
				As[bi] += As [ai];
			}
			offset *= 2;
			}
		// end reduce part
		
		
		
		if ( threadIdx.x == 0){
				Cs[threadIdx.x + n - 1] = As [threadIdx.x + n - 1] ;
				As [threadIdx.x + n - 1] = 0 ; 
		}


		for (int d = 1; d < n; d *= 2){
				offset /= 2 ;
				__syncthreads() ;
				if ( threadIdx.x < d){
					int ai = offset * (2* threadIdx.x +1) -1;
					int bi = offset * (2* threadIdx.x +2) -1;
					float t = As[ai ];
					As[ai] = As[bi ];
					As[bi] += t;
				}
		}

		__syncthreads () ;

		if (threadIdx.x >=1){
				Cs[threadIdx.x - 1] = As[threadIdx.x] ;
		}
		__syncthreads () ;
		cd[x] = Cs[threadIdx.x];			
}









__global__ void scan32(float* ad, float* cd){
		int n =32;
		__shared__ float As[32];
  		__shared__ float Cs[32];
		int offset = 1;
		int x = threadIdx.x + blockIdx.x * blockDim.x;

		As[threadIdx.x] = ad[x];
		// reduce part---------
		for (int d=16 ; d > 0 ; d = d/2){
			__syncthreads() ;
			if ( threadIdx.x < d){
				int ai = offset * (2*  threadIdx.x +1) -1;
				int bi = offset * (2*  threadIdx.x +2) -1;
				As[bi] += As [ai];
			}
			offset *= 2;
			}
		// end reduce part
		
		
		
		if ( threadIdx.x == 0){
				Cs[threadIdx.x + n - 1] = As [threadIdx.x + n - 1] ;
				As [threadIdx.x + n - 1] = 0 ; 
		}


		for (int d = 1; d < n; d *= 2){
				offset /= 2 ;
				__syncthreads() ;
				if ( threadIdx.x < d){
					int ai = offset * (2* threadIdx.x +1) -1;
					int bi = offset * (2* threadIdx.x +2) -1;
					float t = As[ai ];
					As[ai] = As[bi ];
					As[bi] += t;
				}
		}

		__syncthreads () ;

		if (threadIdx.x >=1){
				Cs[threadIdx.x - 1] = As[threadIdx.x] ;
		}
		__syncthreads () ;
		cd[x] = Cs[threadIdx.x];			
}












void scan20(float* a, float* c,int n) {

	float* ad;
	float* cd;
	float* cqd;

  	hipMalloc((void**)&ad, n * sizeof(float));
  	hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 1024 * sizeof(float));

  	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);


	dim3 blocks(1024);
	dim3 threadsPerBlock(1024);
	scanl<<<blocks,threadsPerBlock>>>(ad,cd,cqd);
	scanl<<<1,1024>>>(cqd,cqd,ad);
	add<<<1024,1024>>>(cd,cqd);

	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
  	hipFree(ad);
  	hipFree(cd);
	hipFree(cqd);


}




void scan21(float* a, float* c,int n) {

	float* ad;
	float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, n * sizeof(float));
  	hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 2048* sizeof(float));
	hipMalloc((void**)&cqd1, 2 * sizeof(float));

  	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);


	dim3 blocks(2048);
	dim3 threadsPerBlock(1024);
	scanl<<<blocks,threadsPerBlock>>>(ad,cd,cqd);
	scanl<<<2,1024>>>(cqd,cqd,cqd1);
	scan_s<<<1,1>>>(cqd1,cqd1,2);
	add<<<2,1024>>>(cqd,cqd1);
	add<<<blocks,1024>>>(cd,cqd);


	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
  	hipFree(ad);
  	hipFree(cd);
	hipFree(cqd);
	hipFree(cqd1);



}


void scan22(float* a, float* c,int n) {

	float* ad;
	float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, n * sizeof(float));
  	hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 4096* sizeof(float));
	hipMalloc((void**)&cqd1, 4 * sizeof(float));

  	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);


	dim3 blocks(4096);
	dim3 threadsPerBlock(1024);
	scanl<<<blocks,threadsPerBlock>>>(ad,cd,cqd);
	scanl<<<4,1024>>>(cqd,cqd,cqd1);
	scan_s<<<1,1>>>(cqd1,cqd1,4);
	add<<<4,1024>>>(cqd,cqd1);
	add<<<blocks,1024>>>(cd,cqd);


	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
  	hipFree(ad);
  	hipFree(cd);
	hipFree(cqd);
	hipFree(cqd1);



}





void scan23(float* a, float* c,int n) {

	float* ad;
	float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, n * sizeof(float));
  	hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 8192 * sizeof(float));
	hipMalloc((void**)&cqd1, 8 * sizeof(float));

  	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);


	dim3 blocks(8192);
	dim3 threadsPerBlock(1024);
	scanl<<<blocks,threadsPerBlock>>>(ad,cd,cqd);
	scanl<<<8,1024>>>(cqd,cqd,cqd1);
	scan_s<<<1,1>>>(cqd1,cqd1,8);
	add<<<8,1024>>>(cqd,cqd1);
	add<<<blocks,1024>>>(cd,cqd);


	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
  	hipFree(ad);
  	hipFree(cd);
	hipFree(cqd);
	hipFree(cqd1);



}






void scan24(float* a, float* c,int n) {

	float* ad;
	float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, n * sizeof(float));
  	hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 16384 * sizeof(float));
	hipMalloc((void**)&cqd1, 16 * sizeof(float));

  	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);


	dim3 blocks(16384);
	dim3 threadsPerBlock(1024);
	scanl<<<blocks,threadsPerBlock>>>(ad,cd,cqd);
	scanl<<<16,1024>>>(cqd,cqd,cqd1);
	scan_s<<<1,1>>>(cqd1,cqd1,16);
	add<<<16,1024>>>(cqd,cqd1);
	add<<<blocks,1024>>>(cd,cqd);


	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
  	hipFree(ad);
  	hipFree(cd);
	hipFree(cqd);
	hipFree(cqd1);



}





void scan25(float* a, float* c,int n) {

	float* ad;
	float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, n * sizeof(float));
  	hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 32768 * sizeof(float));
	hipMalloc((void**)&cqd1, 32 * sizeof(float));
  	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);


	dim3 blocks(32768);
	dim3 threadsPerBlock(1024);
	scanl<<<blocks,threadsPerBlock>>>(ad,cd,cqd);
	scanl<<<32,1024>>>(cqd,cqd,cqd1);
	scan32<<<1,32>>>(cqd1,cqd1);
	add<<<32,1024>>>(cqd,cqd1);
	add<<<blocks,1024>>>(cd,cqd);


	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
  	hipFree(ad);
  	hipFree(cd);
	hipFree(cqd);
	hipFree(cqd1);



}






void scan26(float* a, float* c,int n) {

	float* ad;
	float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, n * sizeof(float));
  	hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 65536 * sizeof(float));
	hipMalloc((void**)&cqd1, 64 * sizeof(float));

  	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);
  


	dim3 blocks(32768,2);
	dim3 threadsPerBlock(1024,1);
	scanl<<<blocks,threadsPerBlock>>>(ad,cd,cqd);
	scanl<<<64,1024>>>(cqd,cqd,cqd1);
	scan64<<<1,64>>>(cqd1,cqd1);
	add<<<64,1024>>>(cqd,cqd1);
	add<<<blocks,threadsPerBlock>>>(cd,cqd);

	hipMemcpy(c, cd, n * sizeof(float), hipMemcpyDeviceToHost);
  	hipFree(ad);
  	hipFree(cd);
	hipFree(cqd);
	hipFree(cqd1);



}





void scan27(float* a, float* c,int n) {

	float* ad;
	//float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, n * sizeof(float));
  	//hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 131072 * sizeof(float));
	hipMalloc((void**)&cqd1, 128 * sizeof(float));

  	hipMemcpy(ad, a, n * sizeof(float), hipMemcpyHostToDevice);
  


	dim3 blocks(32768,4);
	dim3 threadsPerBlock(1024,1);
	scanl<<<blocks,threadsPerBlock>>>(ad,ad,cqd);
	scanl<<<128,1024>>>(cqd,cqd,cqd1);
	scan128<<<1,128>>>(cqd1,cqd1);
	add<<<128,1024>>>(cqd,cqd1);
	add<<<blocks,threadsPerBlock>>>(ad,cqd);

	hipMemcpy(c, ad, n * sizeof(float), hipMemcpyDeviceToHost);
  	hipFree(ad);
  	//hipFree(cd);
	hipFree(cqd);
	hipFree(cqd1);



}




void scan28(float* a, float* c,int n) {



	float* ad;
	//float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, (n/2) * sizeof(float));
  	//hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 131072 * sizeof(float));
	hipMalloc((void**)&cqd1, 128 * sizeof(float));

  	hipMemcpy(ad, a, (n/2) * sizeof(float), hipMemcpyHostToDevice);
  


	dim3 blocks(32768,4);
	dim3 threadsPerBlock(1024,1);
	scanl<<<blocks,threadsPerBlock>>>(ad,ad,cqd);
	scanl<<<128,1024>>>(cqd,cqd,cqd1);
	scan128<<<1,128>>>(cqd1,cqd1);
	add<<<128,1024>>>(cqd,cqd1);
	add<<<blocks,threadsPerBlock>>>(ad,cqd);

	hipMemcpy(c, ad, (n/2) * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(cqd);
	hipFree(cqd1);





//-----------------------------------part 2 ------------------------------------


	hipMalloc((void**)&ad, (n/2) * sizeof(float));
  	//hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 131072 * sizeof(float));
	hipMalloc((void**)&cqd1, 128 * sizeof(float));

  	hipMemcpy(ad, a+n/2, (n/2) * sizeof(float), hipMemcpyHostToDevice);
  



	scanl<<<blocks,threadsPerBlock>>>(ad,ad,cqd);
	scanl<<<128,1024>>>(cqd,cqd,cqd1);
	scan128<<<1,128>>>(cqd1,cqd1);
	add<<<128,1024>>>(cqd,cqd1);
	add<<<blocks,threadsPerBlock>>>(ad,cqd);
	add_l<<<blocks,threadsPerBlock>>>(ad,c[n/2-1]);

	hipMemcpy(c+n/2, ad, (n/2) * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(cqd);
	hipFree(cqd1);




}





void scan29(float* a, float* c,int n) {

	float* ad;
	//float* cd;
	float* cqd;
	float* cqd1;

  	hipMalloc((void**)&ad, (n/4) * sizeof(float));
  	//hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 131072 * sizeof(float));
	hipMalloc((void**)&cqd1, 128 * sizeof(float));

  	hipMemcpy(ad, a, (n/4) * sizeof(float), hipMemcpyHostToDevice);
  


	dim3 blocks(32768,4);
	dim3 threadsPerBlock(1024,1);
	scanl<<<blocks,threadsPerBlock>>>(ad,ad,cqd);
	scanl<<<128,1024>>>(cqd,cqd,cqd1);
	scan128<<<1,128>>>(cqd1,cqd1);
	add<<<128,1024>>>(cqd,cqd1);
	add<<<blocks,threadsPerBlock>>>(ad,cqd);

	hipMemcpy(c, ad, (n/4) * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(cqd);
	hipFree(cqd1);





//-----------------------------------part 2 ------------------------------------


	hipMalloc((void**)&ad, (n/4) * sizeof(float));
  	//hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 131072 * sizeof(float));
	hipMalloc((void**)&cqd1, 128 * sizeof(float));

  	hipMemcpy(ad, a+n/4, (n/4) * sizeof(float), hipMemcpyHostToDevice);
  



	scanl<<<blocks,threadsPerBlock>>>(ad,ad,cqd);
	scanl<<<128,1024>>>(cqd,cqd,cqd1);
	scan128<<<1,128>>>(cqd1,cqd1);
	add<<<128,1024>>>(cqd,cqd1);
	add<<<blocks,threadsPerBlock>>>(ad,cqd);
	add_l<<<blocks,threadsPerBlock>>>(ad,c[n/4-1]);

	hipMemcpy(c+n/4, ad, (n/4) * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(cqd);
	hipFree(cqd1);








//-----------------------------------part 3 ------------------------------------


	hipMalloc((void**)&ad, (n/4) * sizeof(float));
  	//hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 131072 * sizeof(float));
	hipMalloc((void**)&cqd1, 128 * sizeof(float));

  	hipMemcpy(ad, a+n/2, (n/4) * sizeof(float), hipMemcpyHostToDevice);
  



	scanl<<<blocks,threadsPerBlock>>>(ad,ad,cqd);
	scanl<<<128,1024>>>(cqd,cqd,cqd1);
	scan128<<<1,128>>>(cqd1,cqd1);
	add<<<128,1024>>>(cqd,cqd1);
	add<<<blocks,threadsPerBlock>>>(ad,cqd);
	add_l<<<blocks,threadsPerBlock>>>(ad,c[n/2-1]);

	hipMemcpy(c+n/2, ad, (n/4) * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(cqd);
	hipFree(cqd1);







//-----------------------------------part 4 ------------------------------------


	hipMalloc((void**)&ad, (n/4) * sizeof(float));
  	//hipMalloc((void**)&cd, n * sizeof(float));
	hipMalloc((void**)&cqd, 131072 * sizeof(float));
	hipMalloc((void**)&cqd1, 128 * sizeof(float));

  	hipMemcpy(ad, a+(n/4)*3, (n/4) * sizeof(float), hipMemcpyHostToDevice);
  



	scanl<<<blocks,threadsPerBlock>>>(ad,ad,cqd);
	scanl<<<128,1024>>>(cqd,cqd,cqd1);
	scan128<<<1,128>>>(cqd1,cqd1);
	add<<<128,1024>>>(cqd,cqd1);
	add<<<blocks,threadsPerBlock>>>(ad,cqd);
	add_l<<<blocks,threadsPerBlock>>>(ad,c[(n/4)*3-1]);

	hipMemcpy(c+(n/4)*3, ad, (n/4) * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(cqd);
	hipFree(cqd1);



}










void gpuKernel(float* a, float* c,int n) {

 switch (n) {
  case 1048576:
    scan20(a,c,n);
    break;

  case 2097152:
    scan21(a,c,n);
    break;

  case 4194304:
    scan22(a,c,n);
    break;

  case 8388608:
    scan23(a,c,n);
    break;

  case 16777216:
    scan24(a,c,n);
    break;

  case 33554432:
    scan25(a,c,n);
   break;

  case 67108864:
    scan26(a,c,n);
    break;

  case 134217728:
    scan27(a,c,n);
    break;


  case 268435456:
    scan28(a,c,n);
    break;


  case 536870912:
    scan29(a,c,n);
    break;
  }
 	

}